#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2019- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_force_sender_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpueventsynchronizer.h"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

/*! \brief Create PME-PP GPU communication object */
PmeForceSenderGpu::Impl::Impl(GpuEventSynchronizer*  pmeForcesReady,
                              MPI_Comm               comm,
                              const DeviceContext&   deviceContext,
                              gmx::ArrayRef<PpRanks> ppRanks) :
    pmeForcesReady_(pmeForcesReady),
    comm_(comm),
    ppRanks_(ppRanks),
    ppCommStream_(ppRanks.size()),
    ppCommEvent_(ppRanks.size()),
    ppCommEventRecorded_(ppRanks.size()),
    deviceContext_(deviceContext),
    pmeRemoteCpuForcePtr_(ppRanks.size()),
    pmeRemoteGpuForcePtr_(ppRanks.size())
{
    // Create streams and events to manage pushing of force buffers to remote PP ranks
    std::unique_ptr<DeviceStream>         stream;
    std::unique_ptr<GpuEventSynchronizer> event;
    size_t                                i = 0;
    for (i = 0; i < ppRanks_.size(); i++)
    {
        stream = std::make_unique<DeviceStream>(deviceContext_, DeviceStreamPriority::High, false);
        ppCommStream_[i] = std::move(stream);
        event            = std::make_unique<GpuEventSynchronizer>();
        ppCommEvent_[i]  = std::move(event);
    }
    stageThreadMpiGpuCpuComm_ = (getenv("GMX_ENABLE_STAGED_GPU_TO_CPU_PMEPP_COMM") != nullptr);
}

PmeForceSenderGpu::Impl::~Impl() = default;

/*! \brief Sets location of force to be sent to each PP rank  */
void PmeForceSenderGpu::Impl::setForceSendBuffer(DeviceBuffer<Float3> d_f)
{

    // Need to send address to PP rank only for thread-MPI as PP rank pulls
    // data using cudamemcpy
    if (!GMX_THREAD_MPI)
    {
        return;
    }

#if GMX_MPI

    if (localForcePtr_.empty())
    {
        localForcePtr_.resize(ppRanks_.size());
    }
    int ind_start = 0;
    int ind_end   = 0;
    int i         = 0;
    for (const auto& receiver : ppRanks_)
    {
        ind_start = ind_end;
        ind_end   = ind_start + receiver.numAtoms;

        if (receiver.numAtoms > 0)
        {
            localForcePtr_[i] = &d_f[ind_start];
            // NOLINTNEXTLINE(bugprone-sizeof-expression)
            MPI_Recv(&pmeRemoteGpuForcePtr_[i], sizeof(float3*), MPI_BYTE, receiver.rankId, 0, comm_, MPI_STATUS_IGNORE);
            // NOLINTNEXTLINE(bugprone-sizeof-expression)
            MPI_Recv(&pmeRemoteCpuForcePtr_[i], sizeof(float3*), MPI_BYTE, receiver.rankId, 0, comm_, MPI_STATUS_IGNORE);
            // Send address of event and associated flag to PP rank, to allow remote enqueueing
            // NOLINTNEXTLINE(bugprone-sizeof-expression)
            MPI_Send(&ppCommEvent_[i], sizeof(GpuEventSynchronizer*), MPI_BYTE, receiver.rankId, 0, comm_);

            std::atomic<bool>* tmpPpCommEventRecordedPtr =
                    reinterpret_cast<std::atomic<bool>*>(&(ppCommEventRecorded_[i]));
            tmpPpCommEventRecordedPtr->store(false, std::memory_order_release);
            // NOLINTNEXTLINE(bugprone-sizeof-expression)
            MPI_Send(&tmpPpCommEventRecordedPtr, sizeof(std::atomic<bool>*), MPI_BYTE, receiver.rankId, 0, comm_);
        }
        i++;
    }

#else
    GMX_UNUSED_VALUE(d_f);
#endif
}


/*! \brief Send PME synchronizer directly using CUDA memory copy */
void PmeForceSenderGpu::Impl::sendFToPpCudaDirect(int ppRank, int numAtoms, bool sendForcesDirectToPpGpu)
{

    GMX_ASSERT(GMX_THREAD_MPI, "sendFToPpCudaDirect is expected to be called only for Thread-MPI");


#if GMX_MPI
    float3* pmeRemoteForcePtr = (sendForcesDirectToPpGpu || stageThreadMpiGpuCpuComm_)
                                        ? pmeRemoteGpuForcePtr_[ppRank]
                                        : pmeRemoteCpuForcePtr_[ppRank];

    pmeForcesReady_->enqueueWaitEvent(*ppCommStream_[ppRank]);

    // Push data to remote GPU's memory
    hipError_t stat = hipMemcpyAsync(pmeRemoteForcePtr,
                                       localForcePtr_[ppRank],
                                       numAtoms * sizeof(rvec),
                                       hipMemcpyDefault,
                                       ppCommStream_[ppRank]->stream());
    CU_RET_ERR(stat, "hipMemcpyAsync on Recv from PME CUDA direct data transfer failed");

    if (stageThreadMpiGpuCpuComm_ && !sendForcesDirectToPpGpu)
    {
        // Perform local D2H (from remote GPU memory to remote PP rank's CPU memory)
        // to finalize staged data transfer
        stat = hipMemcpyAsync(pmeRemoteCpuForcePtr_[ppRank],
                               pmeRemoteGpuForcePtr_[ppRank],
                               numAtoms * sizeof(rvec),
                               hipMemcpyDefault,
                               ppCommStream_[ppRank]->stream());
        CU_RET_ERR(stat, "hipMemcpyAsync on local device to host transfer of PME forces failed");
    }

    ppCommEvent_[ppRank]->markEvent(*ppCommStream_[ppRank]);
    std::atomic<bool>* tmpPpCommEventRecordedPtr =
            reinterpret_cast<std::atomic<bool>*>(&(ppCommEventRecorded_[ppRank]));
    tmpPpCommEventRecordedPtr->store(true, std::memory_order_release);
#else
    GMX_UNUSED_VALUE(ppRank);
    GMX_UNUSED_VALUE(numAtoms);
#endif
}

/*! \brief Send PME data directly using GPU-aware MPI */
void PmeForceSenderGpu::Impl::sendFToPpCudaMpi(DeviceBuffer<RVec> sendbuf,
                                               int                offset,
                                               int                numBytes,
                                               int                ppRank,
                                               MPI_Request*       request)
{
    GMX_ASSERT(GMX_LIB_MPI, "sendFToPpCudaMpi is expected to be called only for Lib-MPI");

#if GMX_MPI
    // if using GPU direct comm with GPU-aware MPI, make sure forces are ready on device
    // before sending it to PP ranks
    pmeForcesReady_->waitForEvent();

    MPI_Isend(sendbuf[offset], numBytes, MPI_BYTE, ppRank, 0, comm_, request);

#else
    GMX_UNUSED_VALUE(sendbuf);
    GMX_UNUSED_VALUE(offset);
    GMX_UNUSED_VALUE(numBytes);
    GMX_UNUSED_VALUE(ppRank);
    GMX_UNUSED_VALUE(request);
#endif
}

PmeForceSenderGpu::PmeForceSenderGpu(GpuEventSynchronizer*  pmeForcesReady,
                                     MPI_Comm               comm,
                                     const DeviceContext&   deviceContext,
                                     gmx::ArrayRef<PpRanks> ppRanks) :
    impl_(new Impl(pmeForcesReady, comm, deviceContext, ppRanks))
{
}

PmeForceSenderGpu::~PmeForceSenderGpu() = default;


void PmeForceSenderGpu::setForceSendBuffer(DeviceBuffer<RVec> d_f)
{
    impl_->setForceSendBuffer(d_f);
}

void PmeForceSenderGpu::sendFToPpCudaMpi(DeviceBuffer<RVec> sendbuf,
                                         int                offset,
                                         int                numBytes,
                                         int                ppRank,
                                         MPI_Request*       request)
{
    impl_->sendFToPpCudaMpi(sendbuf, offset, numBytes, ppRank, request);
}

void PmeForceSenderGpu::sendFToPpCudaDirect(int ppRank, int numAtoms, bool sendForcesDirectToPpGpu)
{
    impl_->sendFToPpCudaDirect(ppRank, numAtoms, sendForcesDirectToPpGpu);
}


} // namespace gmx
