#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2020- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \internal \file
 * \brief
 * Runners for tests of CUDA types compatibility.
 *
 * \author Artem Zhmurov <zhmurov@gmail.com>
 */
#include "gmxpre.h"

#include "typecasts_runner.h"

#include <vector>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/typecasts.cuh"
#include "gromacs/hardware/device_information.h"
#include "gromacs/utility/arrayref.h"
#include "gromacs/utility/exceptions.h"
#include "gromacs/utility/stringutil.h"

namespace gmx
{

namespace test
{

/* \brief Perform a component-wise conversion of the float3 vector back to RVec format.
 *
 * This is needed to pass the data back to the CPU testing code for comparison with the initial input.
 *
 * \param[out] rVecOutput    Output data in RVec format for the output.
 * \param[in]  float3Output  Output data in float3 format.
 * \param[in]  numElements   Size of the data buffers.
 */
void inline saveFloat3InRVecFormat(ArrayRef<gmx::RVec> rVecOutput, const float3* float3Output, int numElements)
{
    for (int i = 0; i < numElements; i++)
    {
        rVecOutput[i][XX] = float3Output[i].x;
        rVecOutput[i][YY] = float3Output[i].y;
        rVecOutput[i][ZZ] = float3Output[i].z;
    }
}

void convertRVecToFloat3OnHost(ArrayRef<gmx::RVec> rVecOutput, ArrayRef<const gmx::RVec> rVecInput)
{
    const int numElements = rVecInput.size();

    float3* dataFloat3 = asFloat3(const_cast<RVec*>(rVecInput.data()));

    saveFloat3InRVecFormat(rVecOutput, dataFloat3, numElements);
}

//! Number of CUDA threads in a block.
constexpr static int c_threadsPerBlock = 256;

/*! \brief GPU kernel to perform type conversion on the device.
 *
 * \param[out] gm_float3Output Buffer to write the output into.
 * \param[in]  gm_rVecInput    Input data in RVec format.
 * \param[in]  size            Size of the data buffers.
 *
 */
static __global__ void convertRVecToFloat3OnDevice_kernel(DeviceBuffer<float3> gm_float3Output,
                                                          DeviceBuffer<RVec>   gm_rVecInput,
                                                          const int            size)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIndex < size)
    {
        gm_float3Output[threadIndex] = asFloat3(gm_rVecInput)[threadIndex];
    }
}

void convertRVecToFloat3OnDevice(ArrayRef<gmx::RVec>       h_rVecOutput,
                                 ArrayRef<const gmx::RVec> h_rVecInput,
                                 const TestDevice*         testDevice)
{
    const DeviceContext& deviceContext = testDevice->deviceContext();
    const DeviceStream&  deviceStream  = testDevice->deviceStream();

    setActiveDevice(testDevice->deviceInfo());

    const int numElements = h_rVecInput.size();

    DeviceBuffer<RVec> d_rVecInput;
    allocateDeviceBuffer(&d_rVecInput, numElements, deviceContext);
    copyToDeviceBuffer(
            &d_rVecInput, h_rVecInput.data(), 0, numElements, deviceStream, GpuApiCallBehavior::Sync, nullptr);

    DeviceBuffer<float3> d_float3Output;
    allocateDeviceBuffer(&d_float3Output, numElements * DIM, deviceContext);

    std::vector<float3> h_float3Output(numElements);

    KernelLaunchConfig kernelLaunchConfig;
    kernelLaunchConfig.gridSize[0]      = (numElements + c_threadsPerBlock - 1) / c_threadsPerBlock;
    kernelLaunchConfig.blockSize[0]     = c_threadsPerBlock;
    kernelLaunchConfig.blockSize[1]     = 1;
    kernelLaunchConfig.blockSize[2]     = 1;
    kernelLaunchConfig.sharedMemorySize = 0;

    auto       kernelPtr  = convertRVecToFloat3OnDevice_kernel;
    const auto kernelArgs = prepareGpuKernelArguments(
            kernelPtr, kernelLaunchConfig, &d_float3Output, &d_rVecInput, &numElements);
    launchGpuKernel(kernelPtr,
                    kernelLaunchConfig,
                    deviceStream,
                    nullptr,
                    "convertRVecToFloat3OnDevice_kernel",
                    kernelArgs);

    copyFromDeviceBuffer(
            h_float3Output.data(), &d_float3Output, 0, numElements, deviceStream, GpuApiCallBehavior::Sync, nullptr);

    saveFloat3InRVecFormat(h_rVecOutput, h_float3Output.data(), numElements);

    freeDeviceBuffer(&d_rVecInput);
    freeDeviceBuffer(&d_float3Output);
}

} // namespace test
} // namespace gmx
