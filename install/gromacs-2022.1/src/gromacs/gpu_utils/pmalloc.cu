/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2012- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \internal \file
 *  \brief Define functions for host-side memory handling when using CUDA devices.
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */

#include "gmxpre.h"

#include "pmalloc.h"

#include <stdlib.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/utility/cstringutil.h"

/*! Allocates nbytes of page-locked memory.
 *  This memory should always be freed using pfree (or with the page-locked
 *  free functions provied by the CUDA library).
 */
void pmalloc(void** h_ptr, size_t nbytes)
{
    hipError_t stat;
    char        strbuf[STRLEN];
    int         flag = hipHostMallocDefault;

    if (nbytes == 0)
    {
        *h_ptr = nullptr;
        return;
    }

    gmx::ensureNoPendingDeviceError("Could not allocate page-locked memory.");

    stat = hipHostMalloc(h_ptr, nbytes, flag);
    sprintf(strbuf, "hipHostMalloc of size %d bytes failed", static_cast<int>(nbytes));
    CU_RET_ERR(stat, strbuf);
}

/*! Frees page locked memory allocated with pmalloc.
 *  This function can safely be called also with a pointer to a page-locked
 *  memory allocated directly with CUDA API calls.
 */
void pfree(void* h_ptr)
{
    hipError_t stat;

    if (h_ptr == nullptr)
    {
        return;
    }

    gmx::ensureNoPendingDeviceError("Could not free page-locked memory.");

    stat = hipHostFree(h_ptr);
    CU_RET_ERR(stat, "hipHostFree failed");
}
