#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2020- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements GPU Force Reduction using CUDA
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_mdlib
 */

#include "gmxpre.h"

#include "gpuforcereduction_impl_internal.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/typecasts.cuh"
#include "gromacs/gpu_utils/vectype_ops.cuh"

namespace gmx
{

constexpr static int c_threadsPerBlock = 128;

template<bool addRvecForce, bool accumulateForce>
static __global__ void reduceKernel(const float3* __restrict__ gm_nbnxmForce,
                                    const float3* __restrict__ rvecForceToAdd,
                                    float3*    gm_fTotal,
                                    const int* gm_cell,
                                    const int  numAtoms)
{

    // map particle-level parallelism to 1D CUDA thread and block index
    const int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // perform addition for each particle
    if (threadIndex < numAtoms)
    {

        float3* gm_fDest = &gm_fTotal[threadIndex];
        float3  temp;

        // Accumulate or set nbnxm force
        if (accumulateForce)
        {
            temp = *gm_fDest;
            temp += gm_nbnxmForce[gm_cell[threadIndex]];
        }
        else
        {
            temp = gm_nbnxmForce[gm_cell[threadIndex]];
        }

        if (addRvecForce)
        {
            temp += rvecForceToAdd[threadIndex];
        }

        *gm_fDest = temp;
    }
}

void launchForceReductionKernel(int                        numAtoms,
                                int                        atomStart,
                                bool                       addRvecForce,
                                bool                       accumulate,
                                const DeviceBuffer<Float3> d_nbnxmForceToAdd,
                                const DeviceBuffer<Float3> d_rvecForceToAdd,
                                DeviceBuffer<Float3>       d_baseForce,
                                DeviceBuffer<int>          d_cell,
                                const DeviceStream&        deviceStream)
{
    float3* d_baseForcePtr      = &(asFloat3(d_baseForce)[atomStart]);
    float3* d_nbnxmForcePtr     = asFloat3(d_nbnxmForceToAdd);
    float3* d_rvecForceToAddPtr = &(asFloat3(d_rvecForceToAdd)[atomStart]);

    // Configure and launch kernel
    KernelLaunchConfig config;
    config.blockSize[0]     = c_threadsPerBlock;
    config.blockSize[1]     = 1;
    config.blockSize[2]     = 1;
    config.gridSize[0]      = ((numAtoms + 1) + c_threadsPerBlock - 1) / c_threadsPerBlock;
    config.gridSize[1]      = 1;
    config.gridSize[2]      = 1;
    config.sharedMemorySize = 0;

    auto kernelFn = addRvecForce
                            ? (accumulate ? reduceKernel<true, true> : reduceKernel<true, false>)
                            : (accumulate ? reduceKernel<false, true> : reduceKernel<false, false>);

    const auto kernelArgs = prepareGpuKernelArguments(
            kernelFn, config, &d_nbnxmForcePtr, &d_rvecForceToAddPtr, &d_baseForcePtr, &d_cell, &numAtoms);

    launchGpuKernel(kernelFn, config, deviceStream, nullptr, "Force Reduction", kernelArgs);
}

} // namespace gmx
